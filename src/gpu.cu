#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <math.h>
#include <numeric>
#include <fstream>

__global__ void
monte_carlo_kernel(float* A_d, double strike_price, double initial_stock_price,
                   double time_to_maturity_years, double risk_free_rate,
                   double volatility, double number_of_simulations,
                   double number_of_time_steps_per_path,
                   double simulations_per_thread,
                   unsigned long seed
                ) {
    auto thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (thread_id >= number_of_simulations)
        return;

    hiprandState local_state;
    hiprand_init(seed, thread_id, 0, &local_state);

    auto delta_t =
        time_to_maturity_years / number_of_time_steps_per_path; // Time step
    auto simulated_price_at_t =
        initial_stock_price; // Start with initial stock price

    for (int i = 0; i < number_of_time_steps_per_path; i++) {
        // Generate a random normal variable
        // introduces randomness to simulate the effects of Brownian Motion in
        // the stock price a positive value is a positive shock, and a negative
        // value is a negative shock to the price the magnitude of the value
        // determines the size of the shock
        // auto random_normal_variable = distribution(rng);
        // auto random_normal_variable = curand_uniform(&local_state);
        auto random_normal_variable = hiprand_normal(&local_state);

        // Use the stochastic differential equation for Geometric Brownian
        // Motion to determine the stock price for each step at each time step,
        // the price is updated based on deterministic drift, and random
        // fluctuation
        simulated_price_at_t *= std::exp(
            (risk_free_rate - 0.5 * volatility * volatility) * delta_t +
            volatility * std::sqrt(delta_t) * random_normal_variable);
    }

    auto payoff = max(simulated_price_at_t - strike_price, 0.0);

    A_d[thread_id] = payoff;
}

void CUDA_CHECK(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__,
               __LINE__);
        exit(EXIT_FAILURE);
    }
}

auto main() -> int {
    auto initial_stock_price = 100.0;
    auto strike_price = 100.0;
    auto time_to_maturity_years = 1.0;
    auto risk_free_rate = 0.05;
    auto volatility = 0.2;

    auto const number_of_simulations = 1 << 20;
    auto number_of_time_steps_per_path = 100;

    auto start = std::chrono::high_resolution_clock::now();

    auto threads_per_block = 1024;
    auto blocks = static_cast<int>(std::ceil(
        static_cast<double>(number_of_simulations) / threads_per_block));

    float* A_d;
    int64_t size = number_of_simulations * sizeof(float);
    auto err_a = hipMalloc((void**) &A_d, size);
    CUDA_CHECK(err_a);

    monte_carlo_kernel<<<blocks, threads_per_block>>>(
        A_d, strike_price, initial_stock_price, time_to_maturity_years,
        risk_free_rate, volatility, number_of_simulations,
        number_of_time_steps_per_path, 1, time(NULL));
    hipDeviceSynchronize();

    float* outputs = new float[number_of_simulations];
    auto err_c = hipMemcpy(outputs, A_d, size, hipMemcpyDeviceToHost);
    CUDA_CHECK(err_c);

    auto average_payoff =
        std::accumulate(outputs, outputs + number_of_simulations, 0.0F) /
        number_of_simulations;

    auto discounted_present_value =
        std::exp(-risk_free_rate * time_to_maturity_years) * average_payoff;

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = end - start;
    std::cout << "Monte Carlo European Call Option Price: "
              << discounted_present_value << std::endl;
    std::cout << "GPU Execution time (ms): "
              << std::chrono::duration_cast<std::chrono::milliseconds>(duration)
                     .count()
              << '\n';
}
